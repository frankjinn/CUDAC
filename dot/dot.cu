
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

#define intMin(a,b) (a<b ? a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
//Ensure that we dont launch too many blocks
const int blocksPerGrid = intMin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void dot(float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    //set the cache values
    cache[cacheIndex] = temp;

    //sychronize threads in this block
    __syncthreads();
    
    //for redection, threadsPerBlock must be a power of 2 because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main(void) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    //allocation memory on the CPU side
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

    //allocate the memory on the GPU
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

    //allocate the memory on the GPU
    hipMalloc((void**)&dev_a,
                N*sizeof(float));
    hipMalloc((void**)&dev_b,
                N*sizeof(float));
    hipMalloc((void**)&dev_partial_c,
                blocksPerGrid*sizeof(float));

    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    };

    hipMemcpy(dev_a, a, N*sizeof(float),
    hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

    c = 0;
    for (int i = 0; i<blocksPerGrid; i++){
        c += partial_c[i];
    }

    #define sum_sqaure(x) (x*(x+1) * (2*x+1)/6)

    printf("Does GPU value %.6g = %.6g?\n", c,
        2*sum_sqaure((float)(N - 1)));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    free(a);
    free(b);
    free(partial_c);
}